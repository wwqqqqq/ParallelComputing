#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define DEFAULT_M 1000
#define DEFAULT_N 1000
#define DEFAULT_L 1000

#define TILE_WIDTH 16

//Total amount of shared memory per block:			49152 bytes
//Maximum number of threads per multiprocessor:		2048
//Maximum number of threads per block :				1024
//Max dimension size of a thread block(x, y, z) :	(1024, 1024, 64)
//Max dimension size of a grid size(x, y, z) :		(2147483647, 65535, 65535)

void multMatrixNoCUDA(const float* a, const float* b, float* c, int m, int n, int l);
__global__  void multMatrixCUDA(const float* a, const float* b, float* c, int m, int n, int l);
__global__  void multMatrixCUDA_tiled(const float* a, const float* b, float* c, int m, int n, int l);
bool InitCUDA();
void generate_matrix(float* mat, int m, int n);

void printMat(const float* mat, int m, int n) {
	printf("\n***********\n");
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++)
			printf("%f\t", mat[i*n + j]);
		printf("\n");
	}
	printf("***********\n");
}



int main(void) {
	int m, n, l;
	m = DEFAULT_M;
	n = DEFAULT_N;
	l = DEFAULT_L;
	//input m, n, and l
	printf("Please input m, n, and l. Input 0 to use default setting.\n");
	int temp = 0;
	scanf("%d", &temp);
	if (temp > 0) {
		m = temp;
		scanf("%d", &temp);
		if (temp > 0) n = temp;
		scanf("%d", &temp);
		if (temp > 0) l = temp;
	}


	if (!InitCUDA()) return 0;

	float *a = (float*)malloc(sizeof(float)*m*n);
	float *b = (float*)malloc(sizeof(float)*n*l);
	float *c = (float*)malloc(sizeof(float)*m*l);
	float *d = (float*)malloc(sizeof(float)*m*l);
	srand((unsigned int)time(NULL));
	generate_matrix(a, m, n);
	generate_matrix(b, n, l);
	clock_t st, ed;

	//CPU
	st = clock();
	multMatrixNoCUDA(a, b, c, m, n, l);
	ed = clock();
	printf("Using CPU time = %lfms\n", (double)(ed - st) / CLOCKS_PER_SEC * 1000);

	//GPU
	float *cuda_a, *cuda_b, *cuda_c, *cuda_d;
	hipMalloc((void**)&cuda_a, sizeof(float)*m*n);
	hipMalloc((void**)&cuda_b, sizeof(float)*n*l);
	hipMalloc((void**)&cuda_c, sizeof(float)*m*l);
	hipMalloc((void**)&cuda_d, sizeof(float)*m*l);
	hipMemcpy(cuda_a, a, sizeof(float)*m*n, hipMemcpyHostToDevice);
	hipMemcpy(cuda_b, b, sizeof(float)*n*l, hipMemcpyHostToDevice);
	int nblocks = (m * l + 255) / 256;
	//calculating...
	st = clock();
	multMatrixCUDA <<< nblocks, 256 >>> (cuda_a, cuda_b, cuda_c, m, n, l);
	hipDeviceSynchronize(); //force CPU to wait until CUDA kernel executing finished
	ed = clock();
	//calculation end

	hipMemcpy(d, cuda_c, sizeof(float)*m*l, hipMemcpyDeviceToHost);
	printf("Using CUDA time = %lfms\n", (double)(ed - st) / CLOCKS_PER_SEC * 1000);
	//test correctness of result
	float error = 0;
	float maxerror = 0;
	int counterror = 0;
	for (int i = 0; i < m * l; i++) {
		float temp = (c[i] > d[i]) ? (c[i] - d[i]) : (d[i] - c[i]);
		if (temp > maxerror) maxerror = temp;
		error += temp;
		if (temp > 1e-5) counterror++;
	}
	if (counterror == 0) {
		printf("Result correct!\n");
	}
	printf("average error = %f, max error = %f\n", error / m / l, maxerror);

	//GPU - tiled algorithm
	dim3 gridSize, blockSize;
	blockSize.x = TILE_WIDTH;
	blockSize.y = TILE_WIDTH; 
	blockSize.z = 1;
	gridSize.x = (m + blockSize.x - 1) / blockSize.x; 
	gridSize.y = (l + blockSize.y - 1) / blockSize.y;
	gridSize.z = 1;
	//calculating...
	st = clock();
	multMatrixCUDA_tiled <<<gridSize, blockSize>>>(cuda_a, cuda_b, cuda_d, m, n, l);
	hipDeviceSynchronize();
	ed = clock();
	//calculation end

	hipMemcpy(d, cuda_d, sizeof(float)*m*l, hipMemcpyDeviceToHost);
	printf("Using CUDA time = %lfms (tiled)\n", (double)(ed - st) / CLOCKS_PER_SEC * 1000);
	//test correctness of result
	error = 0;
	maxerror = 0;
	counterror = 0;
	for (int i = 0; i < m * l; i++) {
		float temp = (c[i] > d[i]) ? (c[i] - d[i]) : (d[i] - c[i]);
		if (temp > maxerror) maxerror = temp;
		error += temp;
		if (temp > 1e-5) counterror++;
	}
	if (counterror == 0) {
		printf("Result correct!\n");
	}
	printf("average error = %f, max error = %f\n", error / m / l, maxerror);

	//end

	/*//print matrix
	printMat(a, m, n);
	printMat(b, n, l);
	printMat(c, m, l);
	printMat(d, m, l);
	*/
	free(a);
	free(b);
	free(c);
	free(d);
	hipFree(cuda_a);
	hipFree(cuda_b);
	hipFree(cuda_c);
	hipFree(cuda_d);

	getchar();
	getchar();
	return 0;
}


bool InitCUDA() {
	int count;
	hipGetDeviceCount(&count);

	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}
	int i;
	for (i = 0; i<count; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		//printDeviceProp(prop);
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) break;
		}
	}
	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}
	hipSetDevice(i);
	return true;
}

void generate_matrix(float* mat, int m, int n) {
	//srand((unsigned int)time(NULL));
	for (int i = 0; i<m; i++)
		for (int j = 0; j<n; j++) {
			//float temp = (float)rand() / RAND_MAX;
			float temp = 0;
			if (rand() % 2 == 1) temp = -temp;
			float temp2 = rand() % 100;
			if (rand() % 2 == 1) temp2 = -temp2;
			mat[i*n + j] = temp2 + temp;
		}
}

__global__  void multMatrixCUDA(const float* a, const float* b, float* c, int m, int n, int l) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int row = idx / l;
	int column = idx % l;

	if (row < m && column < l) {
		float sum = 0;
		for (int i = 0; i < n; i++) {
			sum += a[row * n + i] * b[i * l + column];
		}
		c[idx] = sum;

	}
}

void multMatrixNoCUDA(const float* a, const float* b, float* c, int m, int n, int l) {
	for (int i = 0; i<m; i++) {
		for (int j = 0; j<l; j++) {
			double sum = 0;
			for (int k = 0; k<n; k++) {
				sum += a[i*n + k] * b[k*l + j];
			}
			c[i*l + j] = sum;
		}
	}
}

__global__  void multMatrixCUDA_tiled(const float* a, const float* b, float* c, int m, int n, int l) {
	__shared__ float shared_a[TILE_WIDTH][TILE_WIDTH];
	__shared__ float shared_b[TILE_WIDTH][TILE_WIDTH];
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int row = bx * TILE_WIDTH + tx;
	int col = by * TILE_WIDTH + ty;
	float sum = 0.0;
	for (int i = 0; i*TILE_WIDTH <= n; i++) {
		if (i * TILE_WIDTH + ty < n && row < m)
			shared_a[tx][ty] = a[row * n + i * TILE_WIDTH + ty];
		else shared_a[tx][ty] = 0;
		if (i * TILE_WIDTH + tx < n && col < l)
			shared_b[tx][ty] = b[(i * TILE_WIDTH + tx)*l + col];
		else shared_b[tx][ty] = 0;
		__syncthreads();
		for (int k = 0; k < TILE_WIDTH; k++)
			sum += shared_a[tx][k] * shared_b[k][ty];
		__syncthreads();
	}
	if (row < m && col < l)
		c[row*l + col] = sum;
}
